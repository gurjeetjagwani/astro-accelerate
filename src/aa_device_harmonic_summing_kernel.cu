#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "aa_params.hpp"
#include "aa_device_harmonic_summing_kernel.hpp"

namespace astroaccelerate {

__global__ void simple_harmonic_sum_GPU_kernel(float const* __restrict__ d_input, float *d_output_SNR, ushort *d_output_harmonics, float *d_MSD, int nTimesamples, int nSpectra, int nHarmonics){
  float HS_value, temp_SNR, SNR;
  ushort max_SNR_harmonic;
  int pos;

  // reading 0th harmonic, i.e. fundamental frequency
  pos = blockIdx.x*nSpectra + blockIdx.y*blockDim.x + threadIdx.x;
  if( (blockIdx.y*blockDim.x + threadIdx.x)<nSpectra ){
    HS_value = __ldg(&d_input[pos]);
    SNR = (HS_value - __ldg(&d_MSD[0]))/(__ldg(&d_MSD[1]));
    max_SNR_harmonic = 0;
    
    if(blockIdx.x>0) {
      for(int f=1; f<nHarmonics; f++) {
        if( (blockIdx.x + f*blockIdx.x)<nTimesamples ) {
          pos = (blockIdx.x + f*blockIdx.x)*nSpectra + blockIdx.y*blockDim.x + threadIdx.x;
          HS_value = HS_value + __ldg(&d_input[pos]);
          temp_SNR = (HS_value - __ldg(&d_MSD[f*2]))/(__ldg(&d_MSD[2*f+1])); //assuming white noise 
          if(temp_SNR > SNR) {
            SNR = temp_SNR;
            max_SNR_harmonic = f;
          }
        }
      }
    }
    
    pos = blockIdx.x*nSpectra + blockIdx.y*blockDim.x + threadIdx.x;
    d_output_SNR[pos] = SNR;
    d_output_harmonics[pos] = max_SNR_harmonic;
  }
}

__inline__ __device__ float remove_scalloping_loss(float Xm2, float Xm1, float X0, float Xp1, float Xp2){
    return(X0 + (1.88494/2.0)*(Xm1 + Xp1) + (0.88494/2.0)*(Xm2 + Xp2));
}

template<class const_params>
__inline__ __device__ void get_frequency_bins(float *down, float *step, float const* __restrict__ d_input, int pos){
    if(const_params::remove_scalloping_loss) {
        float Xm2 = d_input[pos - 2];
        float Xm1 = d_input[pos - 1];
        float X0  = d_input[pos + 0];
        float Xp1 = d_input[pos + 1];
        float Xp2 = d_input[pos + 2];
        float Xp3 = d_input[pos + 3];
        (*down) = remove_scalloping_loss(Xm2, Xm1, X0, Xp1, Xp2);
        (*step) = remove_scalloping_loss(Xm1, X0, Xp1, Xp2, Xp3);
    }
    else {
        (*down) = d_input[pos];
        (*step) = d_input[pos + 1];
    }
}

template<class const_params>
__global__ void greedy_harmonic_sum_GPU_kernel(float *d_maxSNR, ushort *d_maxHarmonics, float const* __restrict__ d_input, float const* __restrict__ d_MSD, int nTimesamples, int nDMs, int nHarmonics){
    __shared__ float s_MSD[64];
    float SNR;
    float partial_sum, maxSNR;
    int maxHarmonics;
    
    if(threadIdx.x<nHarmonics) {
        s_MSD[2*threadIdx.x]      = d_MSD[2*threadIdx.x];
        s_MSD[2*threadIdx.x + 1] = d_MSD[2*threadIdx.x + 1];
    }
    
    __syncthreads();
    
    int data_shift=0;
    float data_down = 0, data_step = 0;
    int pos = const_params::nThreads*blockIdx.x + threadIdx.x;
    if( pos > 1 && pos + 3 < nTimesamples) {
        int block_pos = blockIdx.y*nTimesamples + pos;
        get_frequency_bins<const_params>(&data_down, &data_step, d_input, block_pos);
    }
    if( data_step > data_down ) {
        data_shift++;
        partial_sum = data_step;
    }
    else {
        partial_sum = data_down;
    }
    maxSNR = fdividef( (partial_sum - s_MSD[0]), s_MSD[1]);
    maxHarmonics = 0;
    
    for(int h=1; h<nHarmonics; h++){
        int pos = (h+1)*(const_params::nThreads*blockIdx.x + threadIdx.x) + data_shift;
        float data_down = 0, data_step = 0;
        if( pos > 1 && pos + 3 < nTimesamples) {
            int block_pos = blockIdx.y*nTimesamples + pos;
            get_frequency_bins<const_params>(&data_down, &data_step, d_input, block_pos);
        }
        
        if( data_step > data_down ) {
            data_shift++;
            partial_sum = partial_sum + data_step;
        }
        else {
            partial_sum = partial_sum + data_down;
        }
        
        SNR = fdividef( (partial_sum - s_MSD[2*h]), s_MSD[2*h+1] );
        if( SNR > maxSNR ) {
            maxSNR = SNR;
            maxHarmonics = h;
        }
    }
    
    pos = const_params::nThreads*blockIdx.x + threadIdx.x;
    if(pos < nTimesamples) {
        d_maxSNR[blockIdx.y*nTimesamples + pos] = maxSNR;
        d_maxHarmonics[blockIdx.y*nTimesamples + pos] = (ushort) maxHarmonics;
    }
}


template<class const_params>
__inline__ __device__ void get_frequency_bin_value(float *frequency_bin, float const* __restrict__ data, int pos){
    if(const_params::remove_scalloping_loss){
        float Xm2 = data[pos - 2];
        float Xm1 = data[pos - 1];
        float X0  = data[pos];
        float Xp1 = data[pos + 1];
        float Xp2 = data[pos + 2];
        (*frequency_bin) = X0 + (1.88494/2.0)*(Xm1 + Xp1) + (0.88494/2.0)*(Xm2 + Xp2);
    }
    else {
        (*frequency_bin) = data[pos];
    }
}

template<class const_params>
__global__ void three_dimensional_greedy_harmonic_sum(float *d_maxSNR, ushort *d_maxHarmonics, float const* __restrict__ d_input, size_t const N_f, size_t const N_fdot, 
    size_t const N_f_fdot, size_t const max_f_idx, size_t const max_fdot_idx, size_t const max_f_fdot_idx,
    float const* __restrict__ d_MSD, int nTimesamples, int nDMs, size_t const nHarmonics, float const* __restrict__ d_mean, float const* __restrict__ d_stdev)
{
    int pos = blockIdx.x * blockDim.x + threadIdx.x;

    // d_input is flattened, one dimensional array of zw cube, need to calculate 3D indices for bound checking
    size_t f_fdot_idx = (pos / N_f * N_fdot);
    size_t remaining = pos % (N_f * N_fdot);
    size_t fdot_idx = (remaining / N_f);
    size_t f_idx = (remaining % N_f);

    if (fdot_idx < max_fdot_idx && f_idx < max_f_idx && f_fdot_idx < max_f_fdot_idx) {
        double SNR = 0.0;
        size_t harmonic_order = 0;

        size_t fdot_drift = 0;
        size_t f_drift = 0;
        size_t f_fdot_drift = 0;

        // index for storing elements into output arrays, d_maxSNR and d_maxHarmonics
        const size_t output_pos = (max_f_idx * ( max_fdot_idx * f_fdot_idx + fdot_idx) + f_idx);

        size_t f_pos = (N_f * (N_fdot * f_fdot_idx + fdot_idx) + f_idx);
        size_t f_pos_adj = (N_f * (N_fdot * f_fdot_idx + fdot_idx) + f_idx + 1);
        size_t w_pos = (N_f * (N_fdot * f_fdot_idx + (fdot_idx+1)) + f_idx);
        size_t w_pos_adj = (N_f * (N_fdot * f_fdot_idx + (fdot_idx+1)) + f_idx + 1);
        size_t z_pos = (N_f * (N_fdot * (f_fdot_idx+1) + fdot_idx) + f_idx);
        size_t z_pos_adj = (N_f * (N_fdot * (f_fdot_idx+1) + fdot_idx) + f_idx + 1);

        // bound checking to make sure no invalid memory access
        if (z_pos_adj < (N_f * N_fdot * N_f_fdot)) {
            float f_power = d_input[f_pos];
            float f_adj_power = d_input[f_pos_adj];
            float w_power = d_input[w_pos];
            float w_adj_power = d_input[w_pos_adj];
            float z_power = d_input[z_pos];
            float z_adj_power = d_input[z_pos_adj];

            // Find max power out of all six powers
            float hecta[6] = {f_power, f_adj_power, w_power, w_adj_power,z_power, z_adj_power};
            float maxVal = 0.0;
            float *p_maxVal = hecta;
            for (int i = 0; i < 6; ++i) {
                if (hecta[i] > maxVal) {
                    maxVal = hecta[i];
                    p_maxVal = hecta + i;
                }
            }
            if (p_maxVal == hecta + 1) {
                ++f_drift;
            } else if (p_maxVal == hecta + 2) {
                ++fdot_drift;
            } else if (p_maxVal == hecta + 3) {
                ++fdot_drift;
                ++f_drift;
            } else if (p_maxVal == hecta + 4) {
                ++f_fdot_drift;
            } else if (p_maxVal == hecta + 5) {
                ++f_fdot_drift;
                ++fdot_drift;
                ++f_drift;
            }

            double partial_sum = maxVal;
            SNR = fdividef((partial_sum - d_mean[0]), d_stdev[0]);

            // update output arrays
            if (SNR > d_maxSNR[output_pos]) {
                d_maxSNR[output_pos] = SNR;
                d_maxHarmonics[output_pos] = harmonic_order;
            }
            // higher harmonics
            for (size_t h = 1; (h <= nHarmonics) && ( N_f * (N_fdot * (h * N_f_fdot + f_fdot_drift + 1) + (h * fdot_idx + fdot_drift + 1)) + (h * f_idx + f_drift + 1)) < (N_f * N_fdot * N_f_fdot); ++h) {
                f_pos = (h * fdot_idx + fdot_drift) * N_f + (h  * f_idx + f_drift);
                f_pos_adj = (h * fdot_idx + fdot_drift) * N_f + (h  * f_idx + f_drift + 1);
                w_pos = (h * fdot_idx + fdot_drift + 1) * N_f + (h * f_idx + f_drift);
                w_pos_adj = (h  * fdot_idx + fdot_drift + 1) * N_f + (h * f_idx + f_drift + 1);
                z_pos = N_f * (N_fdot * (h * N_f_fdot + f_fdot_drift) + (h * fdot_idx + fdot_drift)) + (h * f_idx + f_drift);
                z_pos_adj = N_f * (N_fdot * (h * N_f_fdot + f_fdot_drift + 1) + (h * fdot_idx + fdot_drift + 1)) + (h * f_idx + f_drift + 1);
        
                f_power = d_input[f_pos];
                f_adj_power = d_input[f_pos_adj];
                w_power = d_input[w_pos];
                w_adj_power = d_input[w_pos_adj];
                z_power = d_input[z_pos];
                z_adj_power = d_input[z_pos_adj];

            float hecta[6] = {f_power, f_adj_power, w_power, w_adj_power,z_power, z_adj_power};
            float maxVal = 0.0;
            float *p_maxVal = hecta;
            for (int i = 0; i < 6; ++i) {
                if (hecta[i] > maxVal) {
                    maxVal = hecta[i];
                    p_maxVal = hecta + i;
                }
            }
            if (p_maxVal == hecta + 1) {
                ++f_drift;
            } else if (p_maxVal == hecta + 2) {
                ++fdot_drift;
            } else if (p_maxVal == hecta + 3) {
                ++fdot_drift;
                ++f_drift;
            } else if (p_maxVal == hecta + 4) {
                ++f_fdot_drift;
            } else if (p_maxVal == hecta + 5) {
                ++f_fdot_drift;
                ++fdot_drift;
                ++f_drift;
            }

            double partial_sum = maxVal;
            SNR = fdividef((partial_sum - d_mean[0]), d_stdev[0]);
                // update output arrays
                if (SNR > d_maxSNR[output_pos]) {
                    d_maxSNR[output_pos] = SNR;
                    d_maxHarmonics[output_pos] = (ushort) h;
                }
            }
        }
    }
}


template<class const_params>
__global__ void presto_plus_harmonic_sum_GPU_kernel(float *d_maxSNR, ushort *d_maxHarmonics, float const* __restrict__ d_input, float const* __restrict__ d_MSD, int nTimesamples, int nDMs, int nHarmonics){
    __shared__ float s_MSD[64];
    float SNR;
    float partial_sum, maxSNR, frequency_bin, fundamental;
    int maxHarmonics, pos;
    
    if(threadIdx.x<nHarmonics) {
        s_MSD[2*threadIdx.x]   = d_MSD[2*threadIdx.x];
        s_MSD[2*threadIdx.x+1] = d_MSD[2*threadIdx.x+1];
    }
    
    __syncthreads();
    
    partial_sum = 0;
    frequency_bin = 0;
    fundamental = 0;
    pos = const_params::nThreads*blockIdx.x + threadIdx.x;
    if( (pos > 1) && (pos + 2) < nTimesamples ) {
        int block_pos = blockIdx.y*nTimesamples + pos;
        get_frequency_bin_value<const_params>(&fundamental, d_input, block_pos);
    }
    partial_sum = fundamental;
    maxSNR = fdividef( (partial_sum - s_MSD[0]), s_MSD[1]);
    maxHarmonics = 0;
    
    if( pos > 1 && (pos + 2) < nTimesamples ) {
        for(int i = 1; i < nHarmonics; i++){ //i + 1 = num. of harmonic added;
            partial_sum = fundamental;
            double fundamental_fraction = ((double) pos)/((double) (i + 1));
            for(int f= 1; f<=i; f++){
                int new_pos = (int) ( ( ((double) f)*fundamental_fraction ) + 0.5 );
                int block_pos = blockIdx.y*nTimesamples + new_pos;
                frequency_bin = 0;
                if( new_pos > 1 && (new_pos + 2) < nTimesamples ) {
                    get_frequency_bin_value<const_params>(&frequency_bin, d_input, block_pos);
                }
                partial_sum = partial_sum + frequency_bin;
            }
            SNR = fdividef( (partial_sum - s_MSD[2*i]), s_MSD[2*i + 1]);
            if(SNR>maxSNR) {
                maxSNR = SNR;
                maxHarmonics = i-1;
            }
        }
    }
    //----------------------------------------------
    
    __syncthreads();
    
    pos = const_params::nThreads*blockIdx.x + threadIdx.x;
    if( pos < nTimesamples ){
        int block_pos = blockIdx.y*nTimesamples + pos;
        d_maxSNR[block_pos] = maxSNR;
        d_maxHarmonics[block_pos] = maxHarmonics;
    }
}


template<class const_params>
__global__ void presto_harmonic_sum_GPU_kernel(float *d_maxSNR, ushort *d_maxHarmonics, float const* __restrict__ d_input, float const* __restrict__ d_MSD, int nTimesamples, int nDMs, int nHarmonicsFactor){
    __shared__ float s_MSD[64];
    float SNR;
    float partial_sum, maxSNR, frequency_bin, fundamental;
    int maxHarmonics, pos;
    
    int nHarmonics = (1<<(nHarmonicsFactor-1));
    if(threadIdx.x<nHarmonics) {
        s_MSD[2*threadIdx.x]   = d_MSD[2*threadIdx.x];
        s_MSD[2*threadIdx.x+1] = d_MSD[2*threadIdx.x+1];
    }
    
    __syncthreads();
    
    maxSNR = -10000;
    maxHarmonics = 0;
    partial_sum = 0;
    pos = const_params::nThreads*blockIdx.x + threadIdx.x;
    for(int i = 0; i < nHarmonicsFactor; i++) {
        int harm = (1<<i);
        
        // 1st Harmonic
        float fundamental_frac = ((double) pos)*(1.0/((double) harm));
        int fundamental_pos = (int) (fundamental_frac + 0.5);
        fundamental = 0;
        if(fundamental_pos > 1 && (fundamental_pos + 2) < nTimesamples){
            int block_pos = blockIdx.y*nTimesamples + fundamental_pos;
            get_frequency_bin_value<const_params>(&fundamental, d_input, block_pos);
        }
        partial_sum = fundamental;
        
        // higher harmonics
        for(int f = 2; f <= harm; f = f + 2){
            int new_pos = (int) ( ((float) f)*fundamental_frac + 0.5);
            frequency_bin = 0;
            if( new_pos > 1 && (new_pos + 2) < nTimesamples ) {
                int block_pos = blockIdx.y*nTimesamples + new_pos;
                get_frequency_bin_value<const_params>(&frequency_bin, d_input, block_pos);
            }
            partial_sum = partial_sum + frequency_bin;
        }
        
        SNR = fdividef( (partial_sum - s_MSD[2*(harm/2)]), s_MSD[2*(harm/2) + 1]);
        if(SNR>maxSNR) {
            maxSNR = SNR;
            maxHarmonics = harm-1;
        }
    }
    //----------------------------------------------
    
    __syncthreads();
    
    pos = const_params::nThreads*blockIdx.x + threadIdx.x;
    if( pos < nTimesamples ){
        int block_pos = blockIdx.y*nTimesamples + pos;
        d_maxSNR[block_pos] = maxSNR;
        d_maxHarmonics[block_pos] = maxHarmonics;
    }
}



//-------------------------------------------------------------------
//------------------------------- Callers ---------------------------
//-------------------------------------------------------------------

  /** \brief Kernel wrapper function for simple_harmonic_sum_GPU_kernel kernel function. */
  void call_kernel_simple_harmonic_sum_GPU_kernel(
      const dim3 &grid_size,
      const dim3 &block_size,
      float const *const d_input,
      float *const d_output_SNR,
      ushort *const d_output_harmonics,
      float *const d_MSD,
      const int &nTimesamples,
      const int &nSpectra,
      const int &nHarmonics
  ) {
    simple_harmonic_sum_GPU_kernel<<<grid_size, block_size>>>(d_input, d_output_SNR, d_output_harmonics, d_MSD, nTimesamples, nSpectra, nHarmonics);
  }

  /** \brief Kernel wrapper function for call_kernel_greedy_harmonic_sum_GPU_kernel kernel function. */
  void call_kernel_greedy_harmonic_sum_GPU_kernel(
      const dim3 &grid_size,
      const dim3 &block_size,
      float const *const d_input,
      float *const d_output_SNR,
      ushort *const d_output_harmonics,
      float *const d_MSD,
      const int &nTimesamples,
      const int &nDMs,
      const int &nHarmonics,
      bool enable_scalloping_loss_removal
  ) {
    if(enable_scalloping_loss_removal) {
      greedy_harmonic_sum_GPU_kernel<HRMS_remove_scalloping_loss><<<grid_size, block_size>>>(
          d_output_SNR,
          d_output_harmonics,
          d_input,
          d_MSD,
          nTimesamples,
          nDMs,
          nHarmonics
      );
    }
    else {
      greedy_harmonic_sum_GPU_kernel<HRMS_normal><<<grid_size, block_size>>>(
          d_output_SNR,
          d_output_harmonics,
          d_input,
          d_MSD,
          nTimesamples,
          nDMs,
          nHarmonics
      );
    }
  }

  /** \brief Kernel wrapper function for presto_harmonic_sum_GPU_kernel kernel function. */
  void call_kernel_presto_plus_harmonic_sum_GPU_kernel(
      const dim3 &grid_size,
      const dim3 &block_size,
      float const *const d_input,
      float *const d_output_SNR,
      ushort *const d_output_harmonics,
      float *const d_MSD,
      const int &nTimesamples,
      const int &nDMs,
      const int &nHarmonics,
      bool enable_scalloping_loss_removal
  ) {
    if(enable_scalloping_loss_removal) {
      presto_plus_harmonic_sum_GPU_kernel<HRMS_remove_scalloping_loss><<<grid_size, block_size>>>(
          d_output_SNR,
          d_output_harmonics,
          d_input,
          d_MSD,
          nTimesamples,
          nDMs,
          nHarmonics
      );
    }
    else {
      presto_plus_harmonic_sum_GPU_kernel<HRMS_normal><<<grid_size, block_size>>>(
          d_output_SNR,
          d_output_harmonics,
          d_input,
          d_MSD,
          nTimesamples,
          nDMs,
          nHarmonics
      );
    }
  }
  
  /** \brief Kernel wrapper function for presto_harmonic_sum_GPU_kernel kernel function. */
  void call_kernel_presto_harmonic_sum_GPU_kernel(
      const dim3 &grid_size,
      const dim3 &block_size,
      float const *const d_input,
      float *const d_output_SNR,
      ushort *const d_output_harmonics,
      float *const d_MSD,
      const int &nTimesamples,
      const int &nDMs,
      const int &nHarmonicsFactor,
      bool enable_scalloping_loss_removal
  ) {
    if(enable_scalloping_loss_removal) {
      presto_harmonic_sum_GPU_kernel<HRMS_remove_scalloping_loss><<<grid_size, block_size>>>(
          d_output_SNR,
          d_output_harmonics,
          d_input,
          d_MSD,
          nTimesamples,
          nDMs,
          nHarmonicsFactor
      );
    }
    else {
      presto_harmonic_sum_GPU_kernel<HRMS_normal><<<grid_size, block_size>>>(
          d_output_SNR,
          d_output_harmonics,
          d_input,
          d_MSD,
          nTimesamples,
          nDMs,
          nHarmonicsFactor
      );
    }
  }
  
} //namespace astroaccelerate




